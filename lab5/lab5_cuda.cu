#include "hip/hip_runtime.h"
/****************************************************************************************/
/* CSE 5441 - Lab assignment 5                                                          */
/*      This is a CUDA program to implements a matrix operation involing very large     */
/*      dimesion (4096 X 4096) matrix using a gpu. The implementation uses a grid of    */
/*      size (256 X 256) and block of size (16 X 16) with each block having its own     */
/*      thread.                                                                         */
/*                                                                                      */
/* Name: Ishan Deep                                                                     */
/* Lname.#: deep.24                                                                     */
/*                                                                                      */
/* Compile using 'nvcc' compiler                                                        */
/* Created on Apr 12 2020                                                               */
/*                                                                                      */
/****************************************************************************************/

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#define N 4096                                  // row/col size of matrix
#define SIZE N*N                                // dimension of matrix
#define MAX_CELL_VALUE 2                        // max value of cells in matrix

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// Method to display any error returned by cuda routines
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Method to initialize a square matrix of size (N X N)
//      -> if setToZero is 1 then cell values will be 0, else it will be set to randomly generated float value
void initMatrix(float *inputMatrix, int setToZero)
{
    srand(time(NULL));                                  // initialize the random number generator
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            inputMatrix[i*N + j] = (!setToZero) ? ((float)rand()/(float)RAND_MAX) * MAX_CELL_VALUE : 0.0;
    }
}

// Method to display a square matrix of size (N X N)
void printMatrix(float *matrix)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%0.1f\t", matrix[i*N + j]);
        
        printf("\n");
    }
}

// Method to compare two a square matrices of size (N X N)
void cmpMatrix(float *firstArr, float *secondArr)
{
    char cell1[20], cell2[20];
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            sprintf(cell1, "%0.1f", firstArr[i*N + j]);
            sprintf(cell2, "%0.1f", secondArr[i*N + j]);
            if (strcmp(cell1, cell2) != 0)
            {
                printf("Two matrices are not equal. Array1=%s and Array2=%s at [%d][%d]\n", cell1, cell2, i, j);
                return;
            }
        }
    }

    printf("Correct answer\n");
}

// Method to perform a operation square matrix of size (N X N) serially.
void doMatrixOperationSerial(float *result, float *inputMatrix)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < N; k++)
                result[i*N + j] += inputMatrix[k*N + i] * inputMatrix[k*N + j];
        }
    }
}

// Method to perform a operation square matrix of size (N X N) parallely using CUDA.
__global__ void doMatrixOperation(float *result, float *inputMatrix, int n)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    for (int k = 0; k < n; k++)
        result[i*n + j] += inputMatrix[k*n + i] * inputMatrix[k*n + j];
}

// main entry point
int main()
{
    float *h_result = (float*)malloc(sizeof(float)*SIZE);                   // host result matrix.
    float *h_resultSerial = (float*)malloc(sizeof(float)*SIZE);             // host result matrix to store result from serial computation.
    float *h_inputMatrix = (float*)malloc(sizeof(float)*SIZE);              // host input matrix on which conputation is performed.
    float *d_result;                                                        // device result matrix.
    float *d_inputMatrix;                                                   // device input matrix on which computation is performed.

    initMatrix(h_result, 1);                                                // init host results matrix to 0s. This is used to store result from cuda computation.
    initMatrix(h_inputMatrix, 0);                                           // init host input matrix to random floating point numbers.
    initMatrix(h_resultSerial, 1);                                          // init host results matrix to 0s. This is used to store result from serial computation.

    gpuErrchk(hipMalloc((void **)&d_result, sizeof(float)*SIZE));          // allocate memory for result matrix on device.
    gpuErrchk(hipMalloc((void **)&d_inputMatrix, sizeof(float)*SIZE));     // allocate memory for input matrix on device.

    // init result and input matrix on device by sending matrix from host to device
    gpuErrchk(hipMemcpy(d_result, h_result, sizeof(float)*SIZE, hipMemcpyHostToDevice));              
    gpuErrchk(hipMemcpy(d_inputMatrix, h_inputMatrix, sizeof(float)*SIZE, hipMemcpyHostToDevice));

    dim3 dimGrid(256, 256);                                                 // set grid layout to (256 X 256)
    dim3 dimBlock(16, 16);                                                  // set block layout to (16 X 16)
    doMatrixOperation<<<dimGrid, dimBlock>>>(d_result, d_inputMatrix, N);   // call kernel
    hipDeviceSynchronize();                                                // Wait for compute device to finish.
    gpuErrchk(hipPeekAtLastError());                                       // check for errors
    gpuErrchk(hipDeviceSynchronize());                                     // check for errors

    // copy final results of matrix operation from device (result array) to host (results array).
    gpuErrchk(hipMemcpy(h_result, d_result, sizeof(float)*SIZE, hipMemcpyDeviceToHost));
    
    doMatrixOperationSerial(h_resultSerial, h_inputMatrix);
    cmpMatrix(h_result, h_resultSerial);
    // printf("Parllel compute:\n");
    // printMatrix(h_result);
    // printf("Serial compute:\n");
    // printMatrix(h_resultSerial);

    return(0);
}